#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <string>

using namespace std;

#include <Windows.h>
#include <windowsx.h>

#define GL_GLEXT_PROTOTYPES

#include <EGL/egl.h>
#include <EGL/eglext.h>
#include <EGL/eglplatform.h>

#include <GLES2/gl2.h>
#include <GLES2/gl2ext.h>

#include <memory>
#include <stdio.h>
#include <stdint.h>
#include <stdlib.h> 
#include <stdbool.h>

#define ERROR_FORMAT(FORMAT, PARAMS) fprintf(stderr, FORMAT, PARAMS); exit(-1);
#define ERROR_TEXT(TEXT) ERROR_FORMAT("%s\n", TEXT)

static HDC eglHDC;
static EGLNativeWindowType eglWindow;
static EGLDisplay eglDisplay;
static EGLContext eglContext;
static EGLSurface eglSurface;

static bool run = false;

static wstring eglWindowClass = L"CudaVoxel";

static LRESULT CALLBACK WndProc(HWND hWnd, UINT message, WPARAM wParam, LPARAM lParam)
{
	switch (message)
	{
		//Destroy / close
		case WM_DESTROY:
		case WM_CLOSE:
		{
			run = false;
			break;
		}

		//Window resize
		case WM_SIZE:
		{
			RECT winRect;
			GetClientRect(hWnd, &winRect);

			POINT topLeft;
			topLeft.x = winRect.left;
			topLeft.y = winRect.top;
			ClientToScreen(hWnd, &topLeft);

			POINT botRight;
			botRight.x = winRect.right;
			botRight.y = winRect.bottom;
			ClientToScreen(hWnd, &botRight);

			//MainResolutionChanged((uint16_t)(botRight.x - topLeft.x), (uint16_t)(botRight.y - topLeft.y));

			break;
		}
	}

	return DefWindowProcW(hWnd, message, wParam, lParam);
}

bool egl_window_create(LONG width, LONG height, wstring title)
{
	WNDCLASSEXW windowClass = { 0 };
	windowClass.cbSize = sizeof(WNDCLASSEXW);
	windowClass.style = CS_OWNDC;
	windowClass.lpfnWndProc = WndProc;
	windowClass.cbClsExtra = 0;
	windowClass.cbWndExtra = 0;
	windowClass.hInstance = GetModuleHandle(NULL);
	windowClass.hIcon = NULL;
	windowClass.hCursor = LoadCursorA(NULL, IDC_ARROW);
	windowClass.hbrBackground = 0;
	windowClass.lpszMenuName = NULL;
	windowClass.lpszClassName = eglWindowClass.c_str();
	if (!RegisterClassExW(&windowClass))
	{
		return false;
	}

	DWORD style = (WS_CAPTION | WS_MINIMIZEBOX | WS_THICKFRAME | WS_MAXIMIZEBOX | WS_SYSMENU);
	DWORD extendedStyle = WS_EX_APPWINDOW;

	RECT sizeRect = { 0, 0, width, height };
	AdjustWindowRectEx(&sizeRect, style, false, extendedStyle);

	width = sizeRect.right - sizeRect.left;
	height = sizeRect.bottom - sizeRect.top;

	//Create the actual window
	eglWindow = CreateWindowExW(extendedStyle, eglWindowClass.c_str(), title.c_str(), style, CW_USEDEFAULT, CW_USEDEFAULT,
		width, height, NULL, NULL,
		GetModuleHandle(NULL), nullptr);

	//Center the window
	HWND   hwndScreen;
	RECT   rectScreen;
	hwndScreen = GetDesktopWindow();
	GetWindowRect(hwndScreen, &rectScreen);

	int posX = ((rectScreen.right - rectScreen.left) / 2 - (width / 2));
	int posY = ((rectScreen.bottom - rectScreen.top) / 2 - (height / 2));
	SetWindowPos(eglWindow, NULL, posX, posY, 0, 0, SWP_SHOWWINDOW | SWP_NOSIZE);

	//Get the handle
	eglHDC = GetDC(eglWindow);
	if (!eglHDC)
	{
		return false;
	}

	return true;
}

void egl_window_cleanup()
{
	if (eglHDC)
	{
		ReleaseDC(eglWindow, eglHDC);
		eglHDC = 0;
	}

	if (eglWindow)
	{
		DestroyWindow(eglWindow);
		eglWindow = 0;
	}

	UnregisterClassW(eglWindowClass.c_str(), NULL);
}

bool egl_init()
{
	const EGLint configAttributes[] =
	{
		EGL_RED_SIZE, 8,
		EGL_GREEN_SIZE, 8,
		EGL_BLUE_SIZE, 8,
		EGL_ALPHA_SIZE, 8,
		EGL_DEPTH_SIZE, 16,
		EGL_STENCIL_SIZE, 0,
		EGL_NONE
	};

	const EGLint surfaceAttributes[] =
	{
		EGL_NONE
	};

	const EGLint contextAttibutes[] =
	{
		EGL_CONTEXT_CLIENT_VERSION, 2,
		EGL_NONE
	};

	const EGLint displayAttributes[] =
	{
		EGL_PLATFORM_ANGLE_TYPE_ANGLE, EGL_PLATFORM_ANGLE_TYPE_D3D11_ANGLE,
		EGL_NONE,
	};

	EGLConfig config = 0;

	// ANGLE: eglGetPlatformDisplayEXT is an alternative to eglGetDisplay. It allows us to specifically request D3D11 instead of D3D9.
	PFNEGLGETPLATFORMDISPLAYEXTPROC eglGetPlatformDisplayEXT = reinterpret_cast<PFNEGLGETPLATFORMDISPLAYEXTPROC>(eglGetProcAddress("eglGetPlatformDisplayEXT"));
	if (!eglGetPlatformDisplayEXT)
	{
		ERROR_TEXT("Failed to get function eglGetPlatformDisplayEXT");
	}

	eglDisplay = eglGetPlatformDisplayEXT(EGL_PLATFORM_ANGLE_ANGLE, eglHDC, displayAttributes);
	if (eglDisplay == EGL_NO_DISPLAY)
	{
		ERROR_TEXT("Failed to get requested EGL display");
		return false;
	}

	if (eglInitialize(eglDisplay, NULL, NULL) == EGL_FALSE)
	{
		ERROR_TEXT("Failed to initialize EGL");
		return false;
	}

	EGLint numConfigs;
	if ((eglChooseConfig(eglDisplay, configAttributes, &config, 1, &numConfigs) == EGL_FALSE) || (numConfigs == 0))
	{
		ERROR_TEXT("Failed to choose first EGLConfig");
		return false;
	}

	eglSurface = eglCreateWindowSurface(eglDisplay, config, eglWindow, surfaceAttributes);
	if (eglSurface == EGL_NO_SURFACE)
	{
		ERROR_TEXT("Failed to create EGL fullscreen surface");
		return false;
	}

	if (eglGetError() != EGL_SUCCESS)
	{
		ERROR_TEXT("eglGetError has reported an error");
		return false;
	}

	eglContext = eglCreateContext(eglDisplay, config, NULL, contextAttibutes);
	if (eglContext == EGL_NO_CONTEXT)
	{
		ERROR_TEXT("Failed to create EGL context");
		return false;
	}

	if (eglMakeCurrent(eglDisplay, eglSurface, eglSurface, eglContext) == EGL_FALSE)
	{
		ERROR_TEXT("Failed to make EGLSurface current");
		return false;
	}

	return true;
}

void gl_setup()
{
	glClearColor(0, 0, 1, 1);
}

void render()
{
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
}

void main_loop()
{
	//Timing
	LONGLONG startTime;
	LONGLONG currentTime;
	LONGLONG frequency;
	double prevElapsedTime = 0.0;

	//Query high precision timer
	LARGE_INTEGER qpfFrequency;
	QueryPerformanceFrequency(&qpfFrequency);
	frequency = qpfFrequency.QuadPart;

	LARGE_INTEGER qpcCurrentTime;
	QueryPerformanceCounter(&qpcCurrentTime);
	startTime = qpcCurrentTime.QuadPart;

	//Loop
	run = true;

	float runTime = 0.0f;

	while (run)
	{
		//Query the timer
		QueryPerformanceCounter(&qpcCurrentTime);
		currentTime = qpcCurrentTime.QuadPart;

		//Calculat the delta time
		double elapsedTime = static_cast<double>(currentTime - startTime) / frequency;
		float deltaTime = (float)(elapsedTime - prevElapsedTime);
		runTime += deltaTime;

		//Call the runtime to do a game loop
		render();

		//Present the new frame
		if (eglSwapBuffers(eglDisplay, eglSurface) != GL_TRUE)
		{
			run = false;
			break;
		}

		//Process Win32 messages
		MSG msg;
		while (PeekMessage(&msg, NULL, 0, 0, PM_REMOVE))
		{
			TranslateMessage(&msg);
			DispatchMessage(&msg);
		}

		//Time keeping
		prevElapsedTime = elapsedTime;
	}
}

#define CUDA_CHECK(STATUS) { hipError_t status = (STATUS); if (status != hipSuccess) { \
	ERROR_FORMAT("CUDA error! %s\n", hipGetErrorName(status)); \
} } \

#define CHUNK_SIZE 16
#define CHUNK_VOXELS (CHUNK_SIZE * CHUNK_SIZE * CHUNK_SIZE)

__global__ void generateChunkKernel(uint8_t* chunkData)
{
    int thread = threadIdx.x;

    int start = thread * (CHUNK_SIZE * CHUNK_SIZE);
    for (int x = start; x < start + (CHUNK_SIZE * CHUNK_SIZE); x++) {
        chunkData[x] = (thread+1);
    }
}

int main()
{
	if (!egl_window_create(1280, 720, L"CUDA Voxel")) 
	{
		return -1;
	}
	if (!egl_init()) 
	{
		return -1;
	}
	ShowWindow(eglWindow, SW_SHOW);

	gl_setup();
	main_loop();

  /*  CUDA_CHECK(hipSetDevice(0));

    uint8_t* chunkDataCPU;
    uint8_t* chunkDataGPU;

    chunkDataCPU = (uint8_t*)malloc(CHUNK_VOXELS);
    memset(chunkDataCPU, 0, CHUNK_VOXELS);

    CUDA_CHECK(hipMalloc((void**)&chunkDataGPU, CHUNK_VOXELS));
    CUDA_CHECK(hipMemcpy(chunkDataGPU, chunkDataCPU, CHUNK_VOXELS, hipMemcpyKind::hipMemcpyHostToDevice));

    generateChunkKernel << <CHUNK_SIZE * CHUNK_SIZE, CHUNK_SIZE >> > (chunkDataGPU);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemcpy(chunkDataCPU, chunkDataGPU, CHUNK_VOXELS, hipMemcpyKind::hipMemcpyDeviceToHost));
    CUDA_CHECK(hipFree(chunkDataGPU));

    free(chunkDataCPU);*/

    CUDA_CHECK(hipDeviceReset());
	egl_window_cleanup();
    return 0;
}
